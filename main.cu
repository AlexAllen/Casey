#include "hip/hip_runtime.h"
#include <fstream>
#include <hiprand/hiprand_kernel.h>
using namespace std;

const char fn[] = "annealing.dump";
const int threadDim = 8;
const int xblocks = 4;
const int yblocks = 4;
const int height = yblocks*threadDim;
const int width = xblocks*threadDim;
const int k1tok3 = 1;
const int arraySize = xblocks*threadDim * yblocks*threadDim + 2; // penultimate cell in array is top boundary, final is bottom boundry

#include "bitsnbobs.cu"

__global__ void monte_kernel(double *nx, double *ny, bool *inp, hiprandState *state, int *hits, double aoa, double iTk,  int offset);
__global__ void energy_kernel(double *nx, double *ny, bool *inp, double *blockEnergies);
__global__ void empty_kernel(int *hits)
{
	hits[blockIdx.x + blockIdx.y*gridDim.x] = 0;
}

int main()
{
	int i, j, loopMax = 5000000;
	double nx[arraySize], ny[arraySize], *dev_nx, *dev_ny;
	bool inp[arraySize], *dev_inp; // is nanoparticle
	char filename[] = "grid.dump";
	double energy, blockEnergies[xblocks*yblocks], *dev_blockEnergies;
	double aoa = PI*0.5, iTk = 0.05;
	int *dev_hits;
	hiprandState *dev_state;
	ofstream out(fn);
	if(!out)
	{
		cout << "Opening files FTL!" << endl;
		return -180;
	}

	// Initialise grid
	gridInit(nx, ny, inp, arraySize);

	// Nanoparticle adders go here

	// Allocate space and copy stuff on/to GPU memory
	danErrHndl( hipMalloc( (void**) &dev_nx, arraySize*sizeof(double) ) );
	danErrHndl( hipMalloc( (void**) &dev_ny, arraySize*sizeof(double) ) );
	danErrHndl( hipMalloc( (void**) &dev_inp, arraySize*sizeof(bool) ) );
	danErrHndl( hipMalloc( (void**) &dev_hits, xblocks*yblocks*sizeof(int)/8 ) );
	danErrHndl( hipMalloc( (void**) &dev_state, (arraySize-2)*sizeof(hiprandState) ) );
	danErrHndl( hipMalloc( (void**) &dev_blockEnergies, xblocks*yblocks*sizeof(double) ) );
	danErrHndl( hipMemcpy( dev_nx, nx, arraySize*sizeof(double), hipMemcpyHostToDevice ) );
	danErrHndl( hipMemcpy( dev_ny, ny, arraySize*sizeof(double), hipMemcpyHostToDevice ) );
	danErrHndl( hipMemcpy( dev_inp, inp, arraySize*sizeof(bool), hipMemcpyHostToDevice ) );

	// Calculate initial energy
	dim3 threads(threadDim, threadDim);
	dim3 blocks(xblocks, yblocks);
	empty_kernel<<<blocks, 1>>>(dev_hits);
	energy_kernel<<<blocks, threads>>>(dev_nx, dev_ny, dev_inp, dev_blockEnergies);

	// Copy back and sum blockEnergies
	danErrHndl( hipMemcpy(blockEnergies, dev_blockEnergies, xblocks*yblocks*sizeof(double), hipMemcpyDeviceToHost) );
	energy=0;
	for(i=0; i<xblocks*yblocks; i++)
	{
		energy+=blockEnergies[i];
	}

	cout << "Initial energy is: " << energy << endl;

	// Watch out for poisonous adders
	dim3 lessBlocks(xblocks/4,yblocks/2);

	cout << "0%";

	int totalHits, hits[xblocks*yblocks/8];

	// The monte carlo loop
	for(j=0;j<loopMax;j++)
	{
		for(i=0;i<8;i++)
		{
			monte_kernel<<<lessBlocks, threads>>>(dev_nx, dev_ny, dev_inp, dev_state, dev_hits, aoa, iTk, intRnd()%8);
		}

		if(!(j%10))
		{
			danErrHndl( hipMemcpy(hits, dev_hits, xblocks*yblocks*sizeof(int)/8, hipMemcpyDeviceToHost) );
			
	        	totalHits = 0;
	        	for(int i=0;i<xblocks*yblocks/8;i++)
	        	{
	        	        totalHits += hits[i];
	       		 	hits[i] = 0;
	        	}

	        	if( aoa > 0.002 ) aoa *= 2 * (double) totalHits / (width*height);
	        	if( aoa > PI*0.5) aoa = 0.5*PI;
	       		if( aoa < 0.002 ) aoa = 0.002;
		}

	        if(!(j%1000))
	        {
	        	iTk *= 1.01;
			if(iTk > 1e7 ) iTk = 1e7;
	        }

		if(!(j%1000))
		{
			energy_kernel<<<blocks, threads>>>(dev_nx, dev_ny, dev_inp, dev_blockEnergies);
			danErrHndl( hipMemcpy(blockEnergies, dev_blockEnergies, xblocks*yblocks*sizeof(double), hipMemcpyDeviceToHost) );
			energy = 0;
       			for(int k=0; k<xblocks*yblocks; k++)
        		{
                		energy += blockEnergies[k];
        		}

			out << j << " " << aoa << " " << iTk << " " << energy << endl;
		}

		danErrHndl( hipMemcpy(dev_hits, hits, xblocks*yblocks*sizeof(int)/8, hipMemcpyHostToDevice) );

		
		if(!(j%100)) cout << "\r" << (double) 100 * j / loopMax << "%                          ";
		
	}

	cout << "\r100%                   " << endl;

	// This is a comment
        energy_kernel<<<blocks, threads>>>(dev_nx, dev_ny, dev_inp, dev_blockEnergies);
	danErrHndl( hipMemcpy(blockEnergies, dev_blockEnergies, xblocks*yblocks*sizeof(double), hipMemcpyDeviceToHost) );
	energy = 0;
	for(i=0; i<xblocks*yblocks; i++)
	{
		energy += blockEnergies[i];
	}

	cout << "Final energy is: " << energy << endl;

	// Get the finished arrays back and dump to file in a dans-gnuplot-script friendly way
	danErrHndl( hipMemcpy(nx, dev_nx, arraySize*sizeof(double), hipMemcpyDeviceToHost));
	danErrHndl( hipMemcpy(ny, dev_ny, arraySize*sizeof(double), hipMemcpyDeviceToHost));
	cout << aoa << " " << iTk << endl;
	outputGrid(nx, ny, inp, filename);
	
	return 0;
}

__global__ void monte_kernel(double *nx, double *ny, bool *inp, hiprandState *state, int *hits, double aoa, double iTk, int offset)
{
	__shared__ int localHits[threadDim*threadDim]; // for local people 

	// calculate cell of interest
	int threadx = threadIdx.x + blockIdx.x * blockDim.x;
	int thready = threadIdx.y + blockIdx.y * blockDim.y;
	int blockID = blockIdx.x + blockIdx.y * gridDim.x;
	int globalID = threadx + thready * blockDim.x * gridDim.x;
	int threadID = threadIdx.x + threadIdx.y * blockDim.x;
	int offsetx = offset%4;
	int offsety = offset/4;
	int x = (thready%2 ? 4*threadx+2 : 4*threadx) + offsetx;
	int y = 2*thready + offsety;
	int index = getIndex(x,y);

	// Don't mess with nanoparticles
	if(inp[index]) return;

	localHits[threadID] = 0;

	double before=0, after=0, dE, rollOfTheDice, angle = PI*aoa*(2*hiprand_uniform(&state[globalID])-1)/180;
	double oldNx = nx[index];
	double oldNy = ny[index];

	before = calcEnergy(x,y,nx,ny);
	before += calcEnergy(x+1,y,nx,ny);
	before += calcEnergy(x-1,y,nx,ny);
	before += calcEnergy(x,y+1,nx,ny);
	before += calcEnergy(x,y-1,nx,ny);

	//rotate director anti-clockwise by angle "angle"
	nx[index] = cos(angle)*oldNx - sin(angle)*oldNy;
	ny[index] = sin(angle)*oldNx + cos(angle)*oldNy;

	after = calcEnergy(x,y,nx,ny);
	after += calcEnergy(x+1,y,nx,ny);
	after += calcEnergy(x-1,y,nx,ny);
	after += calcEnergy(x,y+1,nx,ny);
	after += calcEnergy(x,y-1,nx,ny);

	dE = after - before;

	// Decide the fate of the change
	if(dE>0)
	{
		rollOfTheDice = hiprand_uniform(&state[globalID]);
		if(rollOfTheDice > exp(-dE*iTk)) // then reject change
		{
			nx[index] = oldNx;
			ny[index] = oldNy;
		}
		else localHits[threadID]++;
	}
	else localHits[threadID]++;

	// sum localHits
	__syncthreads();

	int i = blockDim.x * blockDim.y/2;

	while(i>0)
	{
		if(threadID < i) localHits[threadID] += localHits[threadID + i];
		
		__syncthreads();
		i /= 2;
	}

	if(threadID == 0) hits[blockID] += localHits[0];
}
__global__ void energy_kernel(double *nx, double *ny, bool *inp, double *blockEnergies)
{
	__shared__ double energy[threadDim*threadDim];
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int id = threadIdx.x + threadIdx.y * blockDim.x;
	int blockID = blockIdx.x + blockIdx.y * gridDim.x;
	int i = blockDim.x * blockDim.y / 2;

	energy[id] = calcEnergy(x, y, nx, ny);

	// sum for the block
	__syncthreads();
	
	while(i>0)
	{
		if(id < i ) energy[id] += energy[id + i];
		__syncthreads();
		i/=2;
	}

	if(id==0) blockEnergies[blockID] = energy[0];

}
